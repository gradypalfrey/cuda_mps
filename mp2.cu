#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <string.h> 
#include <stdio.h>

#include <iostream>
#include <memory>
#include <string>

#include <stdlib.h>
#include <time.h>  

using namespace std;

#define BLOCKDIM 16
#define arrLen 256

__global__ void addElement(int a[][arrLen], int b[][arrLen], int c[][arrLen], int length) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int n = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < length && n < length) {
		c[i][n] = a[i][n] + b[i][n];
	}
}

__global__ void addRow(int a[][arrLen], int b[][arrLen], int c[][arrLen], int length) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int n = 0; n < length; n++) {
		if (i < length && n < length)
			c[i][n] = a[i][n] + b[i][n];
	}
}

__global__ void addCol(int a[][arrLen], int b[][arrLen], int c[][arrLen], int length) {
	int n = blockIdx.y * blockDim.y + threadIdx.y;

	for (int i = 0; i < length; i++) {
		if (i < length && n < length)
			c[i][n] = a[i][n] + b[i][n];
	}
}

void printMatrix(int a[][arrLen]) {
	for (int i = 0; i < arrLen; i++) {
		for (int n = 0; n < arrLen; n++) {
			cout << a[i][n] << " ";
		}
		cout << endl;
	}
}

int checkMatrix(int arr1[][arrLen], int arr2[][arrLen]) {
	for (int i = 0; i < arrLen; i++) {
		for (int n = 0; n < arrLen; n++) {
			if (arr1[i][n] != arr2[i][n]) {
				return -1;
			}
		}
	}
    return 1;
}

tyedef int matrix[];

size_t dsize;

int matrix_addition() {

	int a[arrLen][arrLen];
	int b[arrLen][arrLen];
	int c[arrLen][arrLen];

	srand(time(NULL));

	for (int i = 0; i < arrLen; i++) {
		for (int n = 0; n < arrLen; n++) {
			a[i][n] = rand() % 10 + 1;
			b[i][n] = rand() % 10 + 1;
			c[i][n] = 0;
		}
	}

	int(*pA)[arrLen], (*pB)[arrLen], (*pC)[arrLen];

	hipMalloc((void**)&pA, (arrLen*arrLen)*sizeof(int));
	hipMalloc((void**)&pB, (arrLen*arrLen)*sizeof(int));
	hipMalloc((void**)&pC, (arrLen*arrLen)*sizeof(int));

	hipMemcpy(pA, a, (arrLen*arrLen)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pB, b, (arrLen*arrLen)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pC, c, (arrLen*arrLen)*sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(c, pC, (arrLen*arrLen)*sizeof(int), hipMemcpyDeviceToHost);

	float time = 0;
    hipEvent_t start, end;
	// hipEventCreate(&start);
	// hipEventCreate(&end);
	// hipEventRecord(start);

	// auto cTmp = new int[arrLen][arrLen];


	// for (int i = 0; i < arrLen; i++) {
	// 	for (int n = 0; n < arrLen; n++) {
	// 		cTmp[i][n] = a[i][n] + b[i][n];
	// 	}
	// }

    // hipEventRecord(end);
	// hipEventSynchronize(end);
	// hipEventElapsedTime(&time, start, end);
	// hipEventDestroy(start);
	// hipEventDestroy(end);
    // cout << "CPU Time: " << time << endl;

    // time = 0;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);

	dim3 threadsPerBlock(BLOCKDIM, BLOCKDIM);
	dim3 numBlocks((int)ceil(arrLen / (float)threadsPerBlock.x), (int)ceil(arrLen / (float)threadsPerBlock.y));
	addElement << <numBlocks, threadsPerBlock >> >(pA, pB, pC, arrLen);

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time, start, end);
	hipEventDestroy(start);
	hipEventDestroy(end);
	cout << "GPU Element Time: " << time << endl;

    // hipMemcpy(c, pC, (arrLen*arrLen)*sizeof(float), hipMemcpyDeviceToHost);

    // int rslt = checkMatrix(c, cTmp);

    // if (rslt) {
    //     cout << "Test Passed" << endl;
    // } else {
    //     cout << "Test Failed" << endl;
    // }

    // time = 0;
	// hipEventCreate(&start);
	// hipEventCreate(&end);
	// hipEventRecord(start);

	// addRow << <numBlocks, threadsPerBlock >> >(pA, pB, pC, arrLen);

	// hipEventRecord(end);
	// hipEventSynchronize(end);
	// hipEventElapsedTime(&time, start, end);
	// hipEventDestroy(start);
	// hipEventDestroy(end);
	// cout << "GPU Row Time: " << time << endl;

    // hipMemcpy(c, pC, (arrLen*arrLen)*sizeof(float), hipMemcpyDeviceToHost);

    // rslt = checkMatrix(c, cTmp);

    // if (rslt) {
    //     cout << "Test Passed" << endl;
    // } else {
    //     cout << "Test Failed" << endl;
    // }

    // time = 0;
	// hipEventCreate(&start);
	// hipEventCreate(&end);
	// hipEventRecord(start);

	// addCol << <numBlocks, threadsPerBlock >> >(pA, pB, pC, arrLen);

	// hipEventRecord(end);
	// hipEventSynchronize(end);
	// hipEventElapsedTime(&time, start, end);
	// hipEventDestroy(start);
	// hipEventDestroy(end);
	// cout << "GPU Col Time: " << time << endl;

	// hipMemcpy(c, pC, (arrLen*arrLen)*sizeof(float), hipMemcpyDeviceToHost);

    // rslt = checkMatrix(c, cTmp);

    // if (rslt) {
    //     cout << "Test Passed" << endl;
    // } else {
    //     cout << "Test Failed" << endl;
    // }


	hipFree(pA);
	hipFree(pB);
	hipFree(pC);

	return 0;
}

int main(int argc, char *argv[])
{

	matrix_addition();

	cout << "Done" << endl;

	return 0;
}