#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <string.h> 
#include <stdio.h>

#include <iostream>
#include <memory>
#include <string>

#include <stdlib.h>
#include <time.h>  

using namespace std;

// #define BLOCKDIM 16
#define TILEWIDTH 2
#define arrLen 8

__global__ void tileMult(int *a, int *b, int *c, int length) {
	int row = threadIdx.x + TILEWIDTH * blockIdx.x;
	int col = threadIdx.y + TILEWIDTH * blockIdx.y;
	__shared__ int sharedM[TILEWIDTH][TILEWIDTH];
	__shared__ int sharedN[TILEWIDTH][TILEWIDTH];
	int temp = 0;
	int l = col * arrLen + row;
	for (int k = 0; k < arrLen / TILEWIDTH; ++k) {
		if (row < arrLen && col < arrLen) {
			sharedM[threadIdx.y][threadIdx.x] = a[col*arrLen + (k* TILEWIDTH + threadIdx.x)];
			sharedN[threadIdx.y][threadIdx.x] = b[row + arrLen * (k* TILEWIDTH + threadIdx.y)];
		}
		else {
			sharedM[threadIdx.y][threadIdx.x] = 0;
			sharedN[threadIdx.y][threadIdx.x] = 0;
		}
		__syncthreads();
		for (int h = 0; h < TILEWIDTH; h++) {
			temp += sharedM[threadIdx.y][h] * sharedN[h][threadIdx.x];
			__syncthreads();
		}
	}
	c[l] = temp;

}

void printMatrix(int *arr1) {
	for (int i = 0; i < arrLen; i++) {
		for (int n = 0; n < arrLen; n++) {
			int idx = i + n*arrLen;
			cout << arr1[idx] << " ";
		}
		cout << endl;
	}
}

int checkMatrix(int *arr1, int *arr2) {
	for (int i = 0; i < arrLen*arrLen; i++) {
		if (arr1[i] != arr2[i]) {
			return -1;
		}
	}
	return 1;
}


int matrix_addition() {

	int *a;
	int *b;
	int *c;
	int *cTmp;

	size_t size = arrLen * arrLen * sizeof(int);

	a = (int*)malloc(size);
	b = (int*)malloc(size);
	c = (int*)malloc(size);
	cTmp = (int*)malloc(size);

	srand(time(NULL));

	for (int i = 0; i < arrLen*arrLen; i++) {
		a[i] = rand() % 10 + 1;
		b[i] = rand() % 10 + 1;
		c[i] = 0;
		cTmp[i] = 0;
	}

	int *pA, *pB, *pC;

	hipMalloc((void**)&pA, size);
	hipMalloc((void**)&pB, size);
	hipMalloc((void**)&pC, size);

	hipMemcpy(pA, a, size, hipMemcpyHostToDevice);
	hipMemcpy(pB, b, size, hipMemcpyHostToDevice);
	hipMemcpy(pC, c, size, hipMemcpyHostToDevice);

	cout << "Start CPU" << endl;

	float time = 0;
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);

	for (int i = 0; i < arrLen; i++) {
		for (int n = 0; n < arrLen; n++) {
			for (int j = 0; j < arrLen; j++) {
				cTmp[i + n*arrLen] += a[i + j*arrLen] * b[j + n*arrLen];
			}
		}
	}

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time, start, end);
	hipEventDestroy(start);
	hipEventDestroy(end);
	cout << "CPU Time: " << time << endl;

    printMatrix(cTmp);
	cout << endl;

	time = 0;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);

	dim3 threadsPerBlock(TILEWIDTH, TILEWIDTH);
	dim3 numBlocks((int)ceil(arrLen / (float)TILEWIDTH), (int)ceil(arrLen / (float)TILEWIDTH));
	tileMult << <numBlocks, threadsPerBlock >> >(pA, pB, pC, arrLen);

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time, start, end);
	hipEventDestroy(start);
	hipEventDestroy(end);
	cout << "GPU Element Time: " << time << endl;

	hipMemcpy(c, pC, (arrLen*arrLen)*sizeof(float), hipMemcpyDeviceToHost);

	printMatrix(c);
	cout << endl;

	int rslt = checkMatrix(c, cTmp);
	cout << rslt << endl;

	if (rslt == -1) {
		cout << "Test Failed" << endl;
	}
	else {
		cout << "Test Passed" << endl;
	}
	
	hipFree(pA);
	hipFree(pB);
	hipFree(pC);
	free(a);
	free(b);
	free(c);
	free(cTmp);

	hipDeviceReset();

	return 0;
}

int main(int argc, char *argv[])
{

	matrix_addition();

	cout << "Done" << endl;

	return 0;
}