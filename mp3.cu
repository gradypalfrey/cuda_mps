#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <string.h> 
#include <stdio.h>

#include <iostream>
#include <memory>
#include <string>

#include <stdlib.h>
#include <time.h>  

using namespace std;

#define BLOCKDIM 16
#define arrLen 16

__global__ void multElement(int *a, int *b, int *c, int length) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int n = blockIdx.y * blockDim.y + threadIdx.y;
    for (int j = 0; j < length; j++) {
	    if (i < length && n < length) {
            int idx = i + n*arrLen;
            c[idx] += a[i + j*length] * b[j + n*length];
        }
	}
}

void printMatrix(int *arr1) {
	for (int i = 0; i < arrLen; i++) {
		for (int n = 0; n < arrLen; n++) {
			int idx = i + n*arrLen;
			cout << arr1[idx] << " ";
		}
		cout << endl;
	}
}

int checkMatrix(int *arr1, int *arr2) {
	for (int i = 0; i < arrLen*arrLen; i++) {
		if (arr1[i] != arr2[i]) {
			return -1;
		}
	}
	return 1;
}


int matrix_addition() {

	int *a;
	int *b;
	int *c;
	int *cTmp;

	size_t size = arrLen * arrLen * sizeof(int);

	a = (int*)malloc(size);
	b = (int*)malloc(size);
	c = (int*)malloc(size);
	cTmp = (int*)malloc(size);

	srand(time(NULL));

	for (int i = 0; i < arrLen*arrLen; i++) {
		a[i] = rand() % 10 + 1;
		b[i] = rand() % 10 + 1;
		c[i] = 0;
		cTmp[i] = 0;
	}

	int *pA, *pB, *pC;

    float time1 = 0;
	hipEvent_t start1, end1;
    hipEventCreate(&start1);
	hipEventCreate(&end1);
	hipEventRecord(start1);

	hipMalloc((void**)&pA, size);
	hipMalloc((void**)&pB, size);
	hipMalloc((void**)&pC, size);

	hipMemcpy(pA, a, size, hipMemcpyHostToDevice);
	hipMemcpy(pB, b, size, hipMemcpyHostToDevice);
	hipMemcpy(pC, c, size, hipMemcpyHostToDevice);

    hipEventRecord(end1);
	hipEventSynchronize(end1);
	hipEventElapsedTime(&time1, start1, end1);
	hipEventDestroy(start1);
	hipEventDestroy(end1);
	cout << "Transfer Time: " << time1 << endl;

	cout << "Start CPU" << endl;

	float time = 0;
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);

	for (int i = 0; i < arrLen; i++) {
		for (int n = 0; n < arrLen; n++) {
			for (int j = 0; j < arrLen; j++) {
				cTmp[i + n*arrLen] += a[i + j*arrLen] * b[j + n*arrLen];
			}
		}
	}

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time, start, end);
	hipEventDestroy(start);
	hipEventDestroy(end);
	cout << "CPU Time: " << time << endl;

	time = 0;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start);

	dim3 threadsPerBlock(BLOCKDIM, BLOCKDIM);
	dim3 numBlocks((int)ceil(arrLen / (float)threadsPerBlock.x), (int)ceil(arrLen / (float)threadsPerBlock.y));
	multElement << <numBlocks, threadsPerBlock >> >(pA, pB, pC, arrLen);

	hipEventRecord(end);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time, start, end);
	hipEventDestroy(start);
	hipEventDestroy(end);
	cout << "GPU Element Time: " << time << endl;

	hipMemcpy(c, pC, (arrLen*arrLen)*sizeof(float), hipMemcpyDeviceToHost);

	int rslt = checkMatrix(c, cTmp);
    cout << rslt << endl;

	if (rslt == -1) {
		cout << "Test Failed" << endl;
	}
	else {
		cout << "Test Passed" << endl;
	}

    // printMatrix(a);
    // cout << endl;

    // printMatrix(b);
    // cout << endl;

    // printMatrix(cTmp);
    // cout << endl;

    // printMatrix(c);
    // cout << endl;

	hipFree(pA);
	hipFree(pB);
	hipFree(pC);
	free(a);
	free(b);
	free(c);
	free(cTmp);

	hipDeviceReset();

	return 0;
}

int main(int argc, char *argv[])
{

	matrix_addition();

	cout << "Done" << endl;

	return 0;
}